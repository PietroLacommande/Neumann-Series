#include "hip/hip_runtime.h"
﻿//
//#include "hip/hip_runtime.h"
//#include ""
//#include <hip/device_functions.h>
//
//#include <stdio.h>
//#include <cstdlib> // For rand()
//#include <ctime>   // For seeding rand()
//
//#define tileLength (int)16
//
//
//hipError_t multiplyWithCuda(float* c, const float* a, const float* b, unsigned int size);
//
//void printMatrix(const float* matrix, int rows, int columns, const char* name) {
//    printf("Matrix %s:\n", name);
//    for (int i = 0; i < rows; ++i) {
//        printf("{ "); // Start of row delimiter
//        for (int j = 0; j < columns; ++j) {
//            printf("%.2f", matrix[i * columns + j]); // Print with 2 decimal place
//            if (j < columns - 1) {
//                printf(" "); // Add space between elements
//            }
//        }
//        printf(" }"); // End of row delimiter
//        printf("\n"); // Newline after each row
//    }
//    printf("\n");
//}
//
//__global__ void extractDiagonalAndOffDiagonal(float* A, float* D, float* E, int length) {
//    int linearIdx = blockIdx.x * blockDim.x + threadIdx.x;
//    if (linearIdx < length) {
//        for (int j = 0; j < length; j++) {
//            if (linearIdx == j) {
//                D[linearIdx * length + j] = A[linearIdx * length + j]; // Diagonal element
//                E[linearIdx * length + j] = 0.0f;          // Off-diagonal is zero
//            }
//            else {
//                D[linearIdx * length + j] = 0.0f;          // Diagonal is zero
//                E[linearIdx * length + j] = A[linearIdx * length + j];// Off-diagonal element
//            }
//        }
//    }
//}
//
//__global__ void invertDiagonal(float* D, float* D_inv, int N) {
//    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//    if (idx < N) {
//        float diag = D[idx * N + idx];
//    }
//}
//
//__global__ void computeDInvE(float* D_inv, float* E, float* DinvE, int N) {
//    int row = blockIdx.y * blockDim.y + threadIdx.y;
//    int col = blockIdx.x * blockDim.x + threadIdx.x;
//
//    if (row < N && col < N) {
//        DinvE[row * N + col] = D_inv[row * N + row] * E[row * N + col];
//    }
//}
//
//
//__global__ void computeDInvEDInv(float* DinvE, float* D_inv, float* DinvEDinv, int N) {
//    int row = blockIdx.y * blockDim.y + threadIdx.y;
//    int col = blockIdx.x * blockDim.x + threadIdx.x;
//
//    if (row < N && col < N) {
//        float value = 0.0f;
//        for (int k = 0; k < N; ++k) {
//            value += DinvE[row * N + k] * D_inv[k * N + k];
//        }
//        DinvEDinv[row * N + col] = value;
//    }
//}
//
//
//__global__ void computeDInvE2DInv(float* DinvE, float* D_inv, float* DinvE2Dinv, int N) {
//    int row = blockIdx.y * blockDim.y + threadIdx.y;
//    int col = blockIdx.x * blockDim.x + threadIdx.x;
//
//    if (row < N && col < N) {
//        float value = 0.0f;
//        for (int k = 0; k < N; ++k) {
//            value += DinvE[row * N + k] * DinvE[k * N + col] * D_inv[k * N + k];
//        }
//        DinvE2Dinv[row * N + col] = value;
//    }
//}
//
//__global__ void combineResults(float* D_inv, float* DinvEDinv, float* DinvE2Dinv, float* A_inv, int N) {
//    int row = blockIdx.y * blockDim.y + threadIdx.y;
//    int col = blockIdx.x * blockDim.x + threadIdx.x;
//
//    if (row < N && col < N) {
//        A_inv[row * N + col] = D_inv[row * N + row] - DinvEDinv[row * N + col] + DinvE2Dinv[row * N + col];
//    }
//}
//
//
//
//__global__ void multiplyKernel(float* c, const float* a, const float* b, int length) {
//    int threadY = threadIdx.y;
//    int threadX = threadIdx.x;
//
//    int row = (tileLength * blockIdx.y) + threadY;
//    int column = (tileLength * blockIdx.x) + threadX;
//
//    __shared__ float tileA[tileLength][tileLength];
//    __shared__ float tileB[tileLength][tileLength];
//
//    float sum = 0;
//
//    int numTiles = (length + tileLength - 1) / tileLength;
//
//    for (int t = 0; t < numTiles; t++) {
//        // Data loading
//        if (row < length && (t * tileLength + threadX) < length) {
//            tileA[threadY][threadX] = a[row * length + (t * tileLength + threadX)];
//        }
//        else {
//            tileA[threadY][threadX] = 0.0f;
//        }
//
//        if (column < length && (t * tileLength + threadY) < length) {
//            tileB[threadY][threadX] = b[(t * tileLength + threadY) * length + column];
//        }
//        else {
//            tileB[threadY][threadX] = 0.0f;
//        }
//
//        __syncthreads();
//
//        for (int k = 0; k < tileLength; k++) {
//            sum += tileA[threadY][k] * tileB[k][threadX];
//        }
//
//        __syncthreads();
//    }
//
//    if (row < length && column < length) {
//        c[(row * length) + column] = sum;
//    }
//}
//
//
//
//int main()
//{
//    const int matrixSize = 16;
//    const float* a = createArray(matrixSize, matrixSize);
//    const float* b = createArray(matrixSize, matrixSize);
//    float c[(matrixSize * matrixSize)] = { 0 };
//
//    // Print matrices A and B before computation
//    printMatrix(a, matrixSize, matrixSize, "A");
//    printMatrix(b, matrixSize, matrixSize, "B");
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = multiplyWithCuda(c, a, b, matrixSize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    // Print result matrix C after computation
//    printMatrix(c, matrixSize, matrixSize, "C");
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t multiplyWithCuda(float* c, const float* a, const float* b, unsigned int size)
//{
//    float* dev_a = 0;
//    float* dev_b = 0;
//    float* dev_c = 0;
//    hipError_t cudaStatus;
//
//    if (size > 64) {
//        printf("Taille de la matrice limité à 64 par 64");
//        return hipErrorInvalidValue;
//    }
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, (size * size) * sizeof(float));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, (size * size) * sizeof(float));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, (size * size) * sizeof(float));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, (size * size) * sizeof(float), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//
//    cudaStatus = hipMemcpy(dev_b, b, (size * size) * sizeof(float), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//
//
//    dim3 threadsParBlock(tileLength, tileLength, 1);
//    dim3 nombreDeBlock((size + tileLength - 1) / tileLength, (size + tileLength - 1) / tileLength);
//
//    multiplyKernel << <nombreDeBlock, threadsParBlock >> > (dev_c, dev_a, dev_b, size);
//
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, (size * size) * sizeof(float), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//
//    return cudaStatus;
//}

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib> // For rand()
#include <ctime>   // For seeding rand()
#include <fstream>
#include <sstream>
#include <vector>


#define tileLength 8

// Utility function to print a matrix
void printMatrix(const float* matrix, int rows, int columns, const char* name) {
    printf("Matrix %s:\n", name);
    for (int i = 0; i < rows; ++i) {
        printf("{ "); // Start of row delimiter
        for (int j = 0; j < columns; ++j) {
            printf("%.8f", matrix[i * columns + j]); // Print with 2 decimal place
            if (j < columns - 1) {
                printf(" "); // Add space between elements
            }
        }
        printf(" }"); // End of row delimiter
        printf("\n"); // Newline after each row
    }
    printf("\n");
}

// CUDA kernel to extract diagonal and off-diagonal matrices
__global__ void extractDiagonalAndOffDiagonal(float* A, float* D, float* E, int length) {
    int linearIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIdx < length) {
        for (int j = 0; j < length; j++) {
            if (linearIdx == j) {
                D[linearIdx * length + j] = A[linearIdx * length + j];
                E[linearIdx * length + j] = 0.0f;
            }
            else {
                D[linearIdx * length + j] = 0.0f;
                E[linearIdx * length + j] = A[linearIdx * length + j];
            }
        }
    }
}

// CUDA kernel to invert the diagonal matrix
__global__ void invertDiagonal(float* D, float* D_inv, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float diag = D[idx * N + idx];
        D_inv[idx * N + idx] = (diag != 0.0f) ? 1.0f / diag : 0.0f; // Handle division by zero
    }
}

// CUDA kernel to compute D^{-1}E
__global__ void computeDInvE(float* D_inv, float* E, float* DinvE, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        DinvE[row * N + col] = D_inv[row * N + row] * E[row * N + col];
    }
}

// CUDA kernel to compute D^{-1}ED^{-1}
__global__ void computeDInvEDInv(float* DinvE, float* D_inv, float* DinvEDinv, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0.0f;
        for (int k = 0; k < N; ++k) {
            value += DinvE[row * N + k] * D_inv[k * N + k];
        }
        DinvEDinv[row * N + col] = value;
    }
}

// CUDA kernel to compute (D^{-1}E)^2D^{-1}
__global__ void computeDInvE2DInv(float* DinvE, float* D_inv, float* DinvE2Dinv, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0.0f;
        for (int k = 0; k < N; ++k) {
            value += DinvE[row * N + k] * DinvE[k * N + col] * D_inv[k * N + k];
        }
        DinvE2Dinv[row * N + col] = value;
    }
}

// CUDA kernel to combine results into A^{-1}
__global__ void combineResults(float* D_inv, float* DinvEDinv, float* DinvE2Dinv, float* A_inv, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        A_inv[row * N + col] = D_inv[row * N + row] - DinvEDinv[row * N + col] + DinvE2Dinv[row * N + col];
    }
}

// Host function for matrix inversion using Neumann series
void matrixInversionNeumann(float* A, float* A_inv, int N) {
    size_t size = N * N * sizeof(float);
    float* d_A, * d_D, * d_E, * d_D_inv, * d_DinvE, * d_DinvEDinv, * d_DinvE2Dinv;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_D, size);
    hipMalloc((void**)&d_E, size);
    hipMalloc((void**)&d_D_inv, size);
    hipMalloc((void**)&d_DinvE, size);
    hipMalloc((void**)&d_DinvEDinv, size);
    hipMalloc((void**)&d_DinvE2Dinv, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // Step 1: Extract D and E
    extractDiagonalAndOffDiagonal << <gridSize, blockSize >> > (d_A, d_D, d_E, N);
    hipDeviceSynchronize();

    // Step 2: Compute D^{-1}
    invertDiagonal << <(N + blockSize.x - 1) / blockSize.x, blockSize.x >> > (d_D, d_D_inv, N);
    hipDeviceSynchronize();

    // Step 3: Compute D^{-1}E
    computeDInvE << <gridSize, blockSize >> > (d_D_inv, d_E, d_DinvE, N);
    hipDeviceSynchronize();

    // Step 4: Compute D^{-1}ED^{-1}
    /*computeDInvEDInv << <gridSize, blockSize >> > (d_DinvE, d_D_inv, d_DinvEDinv, N);
    hipDeviceSynchronize();*/
    computeDInvE << <gridSize, blockSize >> > (d_DinvE, d_D_inv, d_DinvEDinv, N);
    hipDeviceSynchronize();

    //// Step 5: Compute (D^{-1}E)^2D^{-1}
    //computeDInvE2DInv << <gridSize, blockSize >> > (d_DinvE, d_D_inv, d_DinvE2Dinv, N);
    //hipDeviceSynchronize();

    //// Step 6: Combine results to compute A^{-1}
    //combineResults << <gridSize, blockSize >> > (d_D_inv, d_DinvEDinv, d_DinvE2Dinv, d_A, N);
    //hipDeviceSynchronize();

    hipMemcpy(A_inv, d_DinvEDinv, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A);
    hipFree(d_D);
    hipFree(d_E);
    hipFree(d_D_inv);
    hipFree(d_DinvE);
    hipFree(d_DinvEDinv);
    hipFree(d_DinvE2Dinv);
}


// Function to read a matrix from a .txt file
void readMatrixFromFile(const char* filename, float* matrix, int rows, int cols) {
    std::ifstream infile(filename);
    if (!infile.is_open()) {
        fprintf(stderr, "Error: Could not open file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    std::string line;
    int row = 0;

    while (std::getline(infile, line) && row < rows) {
        std::istringstream iss(line);
        std::string value;
        int col = 0;

        while (std::getline(iss, value, ',') && col < cols) {
            matrix[row * cols + col] = std::stof(value); // Convert string to float
            col++;
        }
        row++;
    }

    infile.close();
}

int main() {
    const int N = 8; // Matrix size
    float A[N * N];   // Host matrix to hold the input
    float A_inv[N * N]; // Host matrix to hold the result

    // Step 1: Read matrix from file
    const char* filename = "MatrixA.txt";
    readMatrixFromFile(filename, A, N, N);

    // Print the input matrix
    printf("Original Matrix A:\n");
    printMatrix(A, N, N, "A");

    // Step 2: Perform Neumann series inversion
    matrixInversionNeumann(A, A_inv, N);

    // Print the inverse matrix
    printf("Approximated Inverse Matrix A^-1:\n");
    printMatrix(A_inv, N, N, "A^-1");

    return 0;
}
